#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "boidsGPU.h"
#include "boids.h"
#include "SFML/Window.hpp"
#include "SFML/Graphics.hpp"


__host__ void boidsGPU(std::vector<Boids> &boidsObj,sf::RenderWindow& window){
    int blockSize = 256;
    int totalBoids = boidsObj.size();
    int numBlocks = (totalBoids + blockSize - 1) / blockSize;
    int maxBoids = 10000;

    //memory allocate 
    float *boid;
    float *px,*py;//xy accelerations of each boid
    float *vx,*vy;//xy accelerations of each boid
    float *ax,*ay;//xy accelerations of each boid

    //copy values from boidsObj to the 
    
    hipMallocManaged(&boid,totalBoids*sizeof(float));
    hipMallocManaged(&px,totalBoids*sizeof(float));
    hipMallocManaged(&py,totalBoids*sizeof(float));
    hipMallocManaged(&vx,totalBoids*sizeof(float));
    hipMallocManaged(&vy,totalBoids*sizeof(float));
    hipMallocManaged(&ax,totalBoids*sizeof(float));
    hipMallocManaged(&ay,totalBoids*sizeof(float));

    for(int i=0;i<totalBoids;i++){
        px[i] = boidsObj[i].getPosition().x;
        py[i] = boidsObj[i].getPosition().y;
        vx[i] = boidsObj[i].getVelocity().x;
        vy[i] = boidsObj[i].getVelocity().y;
        px[i] = boidsObj[i].getAcceleration().x;
        px[i] = boidsObj[i].getAcceleration().y;
    }

    initBoidIDs <<< numBlocks, blockSize >>> (boidsObj, boid,maxBoids);
    calculateForcesGPU <<< numBlocks, blockSize >>> (px,py,vx,vy,ax,ay,boid);
}

__global__ void initBoidIDs(std::vector<Boids> &boidsObj, float *boids,int maxBoids) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= maxBoids) return;
    boids[i] = i;
}
__global__ void calculateForcesGPU(float *px, float *py, float *vx, float *vy, float *ax, float *ay, float *boid){
    // 3 forces
    //alignment
    //cohesion
    //seperation
    //increment accel using above forces
    //append accel to velo
    //append velo to posi
    //draw using posi

    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

}