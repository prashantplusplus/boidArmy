#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "boidsGPU.h"
#include "SFML/Window.hpp"
#include "SFML/Graphics.hpp"


void boidsGPU(std::vector<Boids> &boidsObj,sf::RenderWindow& window){
    int blockSize = 256;
    int totalBoids = boidsObj.size();
    int numBlocks = (totalBoids + blockSize - 1) / blockSize;
    int maxBoids = 10000;

    //memory allocate 
    float *boid;
    float *ax,*ay;//xy accelerations of each boid
    hipMallocManaged(&boid,totalBoids*sizeof(float));
    hipMallocManaged(&ax,totalBoids*sizeof(float));
    hipMallocManaged(&ay,totalBoids*sizeof(float));

    initBoidIDs <<< numBlocks, blockSize >>> (boidsObj, boid,maxBoids);
    calculateForcesGPU <<< numBlocks, blockSize >>> (ax,ay);
    //calculateforces
    //update()
}

__global__ void initBoidIDs(std::vector<Boids> &boidsObj, float *boids,int maxBoids) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= maxBoids) return;
    boids[i] = i;
}
__global__ 
void calculateForcesGPU(float *ax, float *ay){

}