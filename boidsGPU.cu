#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "boidsGPU.h"
#include "boids.h"
#include "SFML/Window.hpp"
#include "SFML/Graphics.hpp"


__host__ void boidsGPU(std::vector<Boids> &boidsObj,sf::RenderWindow& window){
    int blockSize = 256;
    int totalBoids = boidsObj.size();
    int numBlocks = (totalBoids + blockSize - 1) / blockSize;
    int maxBoids = 10000;

    //memory allocate 
    float *boid;
    float *px,*py;//xy accelerations of each boid
    float *vx,*vy;//xy accelerations of each boid
    float *ax,*ay;//xy accelerations of each boid

    //copy values from boidsObj to the 
    
    hipMallocManaged(&boid,totalBoids*sizeof(float));
    hipMallocManaged(&px,totalBoids*sizeof(float));
    hipMallocManaged(&py,totalBoids*sizeof(float));
    hipMallocManaged(&vx,totalBoids*sizeof(float));
    hipMallocManaged(&vy,totalBoids*sizeof(float));
    hipMallocManaged(&ax,totalBoids*sizeof(float));
    hipMallocManaged(&ay,totalBoids*sizeof(float));

    for(int i=0;i<totalBoids;i++){
        px[i] = boidsObj[i].getPosition().x;
        py[i] = boidsObj[i].getPosition().y;
        vx[i] = boidsObj[i].getVelocity().x;
        vy[i] = boidsObj[i].getVelocity().y;
        px[i] = boidsObj[i].getAcceleration().x;
        px[i] = boidsObj[i].getAcceleration().y;
    }

    initBoidIDs <<< numBlocks, blockSize >>> (boidsObj, boid,maxBoids);
    calculateForcesGPU <<< numBlocks, blockSize >>> (px,py,vx,vy,ax,ay,boid);
    drawBoid(boid,px,py);
}

__global__ void initBoidIDs(std::vector<Boids> &boidsObj, float *boids,int maxBoids) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= maxBoids) return;
    boids[i] = i;
}
__global__ void calculateForcesGPU(float *px, float *py, float *vx, float *vy, float *ax, float *ay, float *boid){
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    // 3 forces
    alignment(ax[i],ay[i],sum,count,1.0);
    cohesion(ax[i],ay[i],sum,count,1.0);
    seperation(ax[i],ay[i],sum,count,1.0)

    vx[i] += ax[i];
    vy[i] += ay[i];
    
    px[i] += vx[i];
    py[i] += py[i];
    

}
__global__ void separation(float ax, float ay, int sum, int count, float power)
{
    sf::Vector2f steer;
    for (int i = 0; i < seperationDiff.size(); i++)
    {
        steer.x += seperationDiff[i].x;
        steer.y += seperationDiff[i].y;
    }

    steer.x /= count;
    steer.y /= count;

    normalize(steer);
    steer.x *= 5;
    steer.y *= 5;
    steer.x -= velocity.x;
    steer.y -= velocity.y;
    steer.x *= power;
    steer.y *= power;
    limit(steer, 7);
    ax = steer.x;
    ay = steer.y;
}
__global__ void alignment(float *ax, float *ay, int sum, int count, float power)
{

    sum.x /= count;
    sum.y /= count;

    sum.x *= 5; // Multiply by maxSpeed
    sum.y *= 5; // Multiply by maxSpeed

    sf::Vector2f steer;
    steer.x = sum.x - velocity.x;
    steer.y = sum.y - velocity.y;

    steer.x *= power;
    steer.y *= power;

    limit(steer, 7);
    ax = steer.x;
    ay = steer.y;
}
__global__ void cohesion(float *ax, float *ay, int sum, int count, float power)
{

    sum.x /= count;
    sum.y /= count;

    sf::Vector2f desired;
    desired.x = sum.x - position.x;
    desired.y = sum.y - position.y;

    //normalize(desired);
    desired.x *= 5;
    desired.y *= 5;

    desired.x -= velocity.x;
    desired.y -= velocity.y;

    desired.x *= power;
    desired.y *= power;

    limit(desired, 7);
    ax = desired.x;
    ay = desired.y;
}

void drawBoids(,float *px, float *py){
    for(int i=0;i<totalBoids;i++){
        boids[i] = i;
        sf::CircleShape circle_boid(3);
        circle_boid.setFillColor(color);
        circle_boid.setPosition(px[i], py[i]);
        window.draw(circle_boid);
    }
}